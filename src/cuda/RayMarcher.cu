#include "hip/hip_runtime.h"
#include "RayMarcher.cuh"

#include "CudaContext.cuh"

#include <surface_indirect_functions.h>

// -------------------------------------------------------------------------
// KERNEL

__global__ void ray_marching(VisualizationSettings settings,
							 hipSurfaceObject_t depth,
							 hipSurfaceObject_t depthAgg,
							 hipSurfaceObject_t* positions,
							 hipSurfaceObject_t normals,
							 size_t width, size_t height)
{
	//surf2Dread<float>(depth, x, y);

	for (size_t i = 50 * 1600; i < 52 * 1600; i++)
		surf2Dwrite<float>(1.0f, positions[0], 4 * (i % width), i / width);

	return;

	const size_t stride = gridDim.x * blockDim.x;

	for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < width * height; tid += stride)
	{
		const int x = tid % width;
		const int y = tid / width;

		// byte-addressed, so multiply x coordinate with 4 * sizeof(float)
		//float val = surf2Dread<float>(depth, sizeof(float) * x, y);

		//surf2Dwrite<float4>(make_float4(val, val, val, val), positions, sizeof(float4) * x, y);

		/*surf2Dwrite<float4>(make_float4(1.0f, 1.0f, 1.0f, 1.0f), positions, 16 * x, y);
		surf2Dwrite<float4>(make_float4(1.0f, 1.0f, 1.0f, 1.0f), normals, 16 * x, y);*/

		//const float z = depth[tid];

		/*positions[tid] = glm::vec4(0);
		normals[tid] = glm::vec4(0);
		if (z == 1.0f) continue;*/

#if 0
		auto& frame = m_Dataset->Frames[m_Settings.Frame];

		const glm::vec3 clip(float(index % m_Width) * m_TwoWidthInv - 1.0f,
							 float(index / m_Width) * m_TwoHeightInv - 1.0f,
							 z);

		const glm::vec4 worldH = m_InvProjectionView * glm::vec4(clip, 1);
		glm::vec3 position = glm::vec3(worldH) / worldH.w;
		const glm::vec3 step = m_Settings.StepSize * glm::normalize(position - m_CameraPosition);

		for (int i = 0; i < m_Settings.MaxSteps; i++)
		{
			// step
			position += step;

#if 0
			// check density grid
			OctreeNode* gridNode = nullptr;

			// skip empty space
			while ((gridNode = frame.QueryDensityGrid(position)) && gridNode->Flag == false)
				position = intersectAABB(position, step, gridNode->Min, gridNode->Max) + step;
#endif

			const std::vector<unsigned int> neighbors =
				m_Dataset->GetNeighborsExt(position, m_Settings.Frame);
			locals->NumNeighborsExt = std::min(neighbors.size(), size_t(MAX_NEIGHBORS));
			locals->NumNeighbors = 0;

			// store neighbor positions
			for (uint32_t i = 0; i < locals->NumNeighborsExt; i++)
			{
				locals->NeighborPositions_AbsExt[i] = frame.m_ParticlesExt[neighbors[i]];

				const glm::vec3 r = locals->NeighborPositions_AbsExt[i] - position;

				if (glm::dot(r, r) < m_Dataset->ParticleRadius * m_Dataset->ParticleRadius)
					locals->NeighborPositions_Rel[locals->NumNeighbors++] = r;
			}

			// compute G and density
			glm::mat3 G;
			float density = 0.0f;

			WPCA(position, locals->NeighborPositions_AbsExt, locals->NumNeighborsExt, G);
			const float detG = glm::determinant(G);

			for (uint32_t i = 0; i < locals->NumNeighbors; i++)
				density += m_AnisotropicKernel.W(G, detG, locals->NeighborPositions_Rel[i]);

			if (density >= m_Settings.IsoDensity)
			{
				// set position
				m_Positions[index] = glm::vec4(position, 1);

				// compute object normal
				glm::vec3 normal(0);

				for (uint32_t i = 0; i < locals->NumNeighbors; i++)
					normal += m_AnisotropicKernel.gradW(G, detG, locals->NeighborPositions_Rel[i]);

				m_Normals[index] = glm::vec4(glm::normalize(normal), 1);

				// break
				i = m_Settings.MaxSteps;
			}
		}

#endif
	}
}

// -------------------------------------------------------------------------

void CudaRayMarcher::Setup(const VisualizationSettings& settings,
						   const hipSurfaceObject_t& depth,
						   const hipSurfaceObject_t& depthAgg,
						   hipSurfaceObject_t* positions,
						   const hipSurfaceObject_t& normals,
						   uint32_t width,
						   uint32_t height)
{
	m_Settings = settings;

	m_Depth = depth;
	m_DepthAgg = depthAgg;
	m_Positions = positions;
	m_Normals = normals;
	m_Width = width;
	m_Height = height;

	m_NumThreadsPerBlock = 32;
	m_NumBlocks = (width * height + m_NumThreadsPerBlock - 1) / m_NumThreadsPerBlock;
}

void CudaRayMarcher::Run()
{
	checkCudaErrors(hipPeekAtLastError());

	// lauch kernel
	ray_marching<<<m_NumBlocks, m_NumThreadsPerBlock, 0, Cuda.m_Stream>>>(
		m_Settings,
		m_Depth,
		m_DepthAgg,
		m_Positions,
		m_Normals,
		m_Width,
		m_Height);

	checkCudaErrors(hipDeviceSynchronize());
	//checkCudaErrors(hipStreamSynchronize(Cuda.m_Stream));
	checkCudaErrors(hipPeekAtLastError());
}
