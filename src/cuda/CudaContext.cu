#include "CudaContext.cuh"

// --------------------------------------------------------------

CudaContext& CudaContext::Instance()
{
    static CudaContext ctx;
    return ctx;
}

// --------------------------------------------------------------

bool CudaContext::Init(const vk::Device& vkdevice, uint8_t* vkDeviceUUID, size_t uuidSize)
{
    m_Device = vkdevice;

    // Set the same device that Vulkan uses
    int device = FindDevice(vkDeviceUUID, uuidSize);
    if (device < 0)
    {
        //SPDLOG_ERROR("Failed to find the Cuda device that Vulkan uses.");
        return false;
    }

    checkCudaErrors(hipSetDevice(device));

    // create stream
    checkCudaErrors(hipStreamCreateWithFlags(&m_Stream, hipStreamNonBlocking));

    return true;
}

CudaImageMemoryImportResult CudaContext::ImportImageMemory(const ExternalImage& extImage)
{
    // create CUDA resource handle
    hipExternalMemoryHandleDesc extMemHandleDesc;
    memset(&extMemHandleDesc, 0, sizeof(extMemHandleDesc));

    extMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    extMemHandleDesc.handle.win32.handle = extImage.GetMemoryHandle();
    extMemHandleDesc.size = extImage.m_MemorySize;

    hipExternalMemory_t extMem;
    checkCudaErrors(hipImportExternalMemory(&extMem, &extMemHandleDesc));

    // map mipmap levels to CUDA array layers
    cudaExternalMemoryMipmappedArrayDesc extMemMipmappedArrayDesc;
    memset(&extMemMipmappedArrayDesc, 0, sizeof(extMemMipmappedArrayDesc));

    hipExtent extent{ extImage.m_Width, extImage.m_Height, 0 /* depth=0 for 2D images */};

    hipChannelFormatDesc formatDesc;
    formatDesc.x = extImage.m_NumChannels > 0 ? extImage.m_BitsPerChannel : 0;
    formatDesc.y = extImage.m_NumChannels > 1 ? extImage.m_BitsPerChannel : 0;
    formatDesc.z = extImage.m_NumChannels > 2 ? extImage.m_BitsPerChannel : 0;
    formatDesc.w = extImage.m_NumChannels > 3 ? extImage.m_BitsPerChannel : 0;
    formatDesc.f = hipChannelFormatKindFloat;

    extMemMipmappedArrayDesc.offset = 0;
    extMemMipmappedArrayDesc.formatDesc = formatDesc;
    extMemMipmappedArrayDesc.extent = extent;
    extMemMipmappedArrayDesc.numLevels = 1;
    extMemMipmappedArrayDesc.flags =
#if 1
        //0
        hipArraySurfaceLoadStore
#else
        hipArraySurfaceLoadStore |
        (
            extImage.m_Usage & vk::ImageUsageFlagBits::eColorAttachment
            ? cudaArrayColorAttachment
            : 0
        )
#endif
        ;

    hipMipmappedArray_t mipmappedArray;
    checkCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(
        &mipmappedArray, extMem, &extMemMipmappedArrayDesc));

    checkCudaErrors(hipMallocMipmappedArray(&mipmappedArray, &formatDesc, extent, 1));

    // for each mipmap level (we use only one)
    hipArray_t mipLevelArray; // the data array for one mip level

    checkCudaErrors(hipGetMipmappedArrayLevel(&mipLevelArray, mipmappedArray, 0 /* level */));

    // create surface
    hipResourceDesc resourceDesc;
    memset(&resourceDesc, 0, sizeof(resourceDesc));
    resourceDesc.resType = hipResourceTypeArray;
    resourceDesc.res.array.array = mipLevelArray;

    hipSurfaceObject_t surfaceObject;
    checkCudaErrors(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));

#if 0
    // create mipmapped array resource
    memset(&resourceDesc, 0, sizeof(resourceDesc));

    resourceDesc.resType = hipResourceTypeMipmappedArray;
    resourceDesc.res.mipmap.mipmap = mipmappedArray;

    // create texture
    hipTextureDesc textureDesc;
    memset(&textureDesc, 0, sizeof(textureDesc));

    // maybe can be false?
    textureDesc.normalizedCoords = true;
    textureDesc.filterMode = hipFilterModePoint;
    textureDesc.mipmapFilterMode = hipFilterModePoint;

    textureDesc.addressMode[0] = hipAddressModeWrap;
    textureDesc.addressMode[1] = hipAddressModeWrap;

    textureDesc.maxMipmapLevelClamp = 0.0f;
    textureDesc.readMode = hipReadModeElementType;

    hipTextureObject_t textureObject;
    checkCudaErrors(hipCreateTextureObject(&textureObject, &resourceDesc,
                                            &textureDesc, NULL));
#endif

#if 0
    static std::vector<float> _d(1600*200*4);
    for (size_t i = 0; i < std::size(_d); i++)
        _d[i] = 1.0f;

    checkCudaErrors(
        hipMemcpyToArray(mipLevelArray, 0, 0, _d.data(), sizeof(float) * _d.size(), hipMemcpyHostToDevice));
#endif

    // allocate space for surface handles on device
    hipSurfaceObject_t* deviceSurfaceObject;

    checkCudaErrors(hipMalloc((void**)&deviceSurfaceObject, sizeof(hipSurfaceObject_t)));
    checkCudaErrors(hipMemcpy(deviceSurfaceObject,
                               &surfaceObject,
                               sizeof(hipSurfaceObject_t),
                               hipMemcpyHostToDevice));

    // return
    CudaImageMemoryImportResult result;
    result.surfaceObject = surfaceObject;
    result.deviceSurfaceObject = deviceSurfaceObject;
    result.externalMemory = extMem;
    //result.textureObject = textureObject;

    return result;
}

void CudaContext::ImportSemaphores(HANDLE semaphoreVK, HANDLE semaphoreCU)
{
    ImportSemaphore(semaphoreVK, &m_ExternalSemaphoreVK);
    ImportSemaphore(semaphoreCU, &m_ExternalSemaphoreCU);
}

void CudaContext::WaitForVulkan()
{
    // wait for m_ExternalSemaphoreVK
    hipExternalSemaphoreWaitParams params;
    memset(&params, 0, sizeof(params));

    params.params.fence.value = 0;
    params.flags = 0;

    checkCudaErrors(
        hipWaitExternalSemaphoresAsync(&m_ExternalSemaphoreVK, &params, 1, m_Stream));
}

void CudaContext::SignalVulkan()
{
    // signal m_ExternalSemaphoreCU
    hipExternalSemaphoreSignalParams params;
    memset(&params, 0, sizeof(params));

    params.params.fence.value = 0;
    params.flags = 0;

    checkCudaErrors(
        hipSignalExternalSemaphoresAsync(&m_ExternalSemaphoreCU, &params, 1, m_Stream));
}

int CudaContext::FindDevice(uint8_t* vkDeviceUUID, size_t uuidSize)
{
    int current_device = 0;
    int device_count = 0;
    int devices_prohibited = 0;

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    if (device_count == 0) {
        fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    // Find the GPU which is selected by Vulkan
    while (current_device < device_count) {
        hipGetDeviceProperties(&deviceProp, current_device);

        if ((deviceProp.computeMode != hipComputeModeProhibited)) {
            // Compare the cuda device UUID with vulkan UUID
            int ret = memcmp((void *)&deviceProp.uuid, vkDeviceUUID, uuidSize);
            if (ret == 0) {
                checkCudaErrors(hipSetDevice(current_device));
                checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
                printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
                       current_device, deviceProp.name, deviceProp.major,
                       deviceProp.minor);

                return current_device;
            }

        } else {
            devices_prohibited++;
        }

        current_device++;
    }

    if (devices_prohibited == device_count) {
        fprintf(stderr,
                "CUDA error:"
                " No Vulkan-CUDA Interop capable GPU found.\n");
        exit(EXIT_FAILURE);
    }

    return -1;
}

void CudaContext::ImportSemaphore(HANDLE handle,
                                  hipExternalSemaphore_t* out)
{
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));

    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    externalSemaphoreHandleDesc.handle.win32.handle = handle;

    externalSemaphoreHandleDesc.flags = 0;

    checkCudaErrors(hipImportExternalSemaphore(out, &externalSemaphoreHandleDesc));
}
